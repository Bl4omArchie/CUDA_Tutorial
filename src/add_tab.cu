#include <iostream>
#include <hip/hip_runtime.h>


/*
effectue une addition d'éléments dans deux tableaux et stocke le résultat dans un troisième tableau
*/



const int N = 10;  // Nombre d'éléments dans les tableaux

// Kernel CUDA pour effectuer l'addition de deux tableaux
__global__ void addArrays(const int* a, const int* b, int* result) {
    int i = threadIdx.x;
    if (i < N) {
        result[i] = a[i] + b[i];
    }
}

int main() {
    int a[N], b[N], result[N];  // Tableaux sur l'hôte
    int *d_a, *d_b, *d_result;  // Tableaux sur le GPU

    // Remplir les tableaux a et b sur l'hôte
    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allouer de la mémoire sur le GPU
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_result, N * sizeof(int));

    // Copier les données depuis l'hôte vers le GPU
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Appeler le kernel CUDA pour effectuer l'addition
    addArrays<<<1, N>>>(d_a, d_b, d_result);

    // Copier le résultat depuis le GPU vers l'hôte
    hipMemcpy(result, d_result, N * sizeof(int), hipMemcpyDeviceToHost);

    // Afficher le résultat
    for (int i = 0; i < N; ++i) {
        std::cout << result[i] << " ";
    }
    std::cout << std::endl;

    // Libérer la mémoire sur le GPU
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
