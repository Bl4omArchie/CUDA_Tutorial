#include <iostream>
#include <hip/hip_runtime.h>

const int N = 4;  // Dimension of square matrices (N x N)

__global__ void matrixMultiply(const float* A, const float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < n && col < n) {
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    float A[N * N], B[N * N], C[N * N];  // Matrices on the host
    float *d_A, *d_B, *d_C;  		 // Matrices on the GPU

    // Fill matrices A and B on the host
    for (int i = 0; i < N * N; ++i) {
        A[i] = static_cast<float>(i);
        B[i] = static_cast<float>(i * 2);
    }

    // Copy data from the host to the GPU 
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    // Copy data from the host to the GPU
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Define thread block and grid dimensions
    dim3 threadsPerBlock(N, N);
    dim3 numBlocks(1, 1);

    // Call the CUDA kernel for matrix multiplication
    matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy the result from the GPU to the host
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Display the result matrix
    std::cout << "Result matrix C:" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free memory on the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
