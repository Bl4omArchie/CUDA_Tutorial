
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mul(int *x, int *y, int *z) {
    *z = (*x) * (*y);
}

int main(void) {
    int x, y, z;
    x = 3981;
    y = 713;

    int *g_x, *g_y, *g_z;
    int size = sizeof(int);

    //Allocation and transfert to the GPU
    hipMalloc((void **) &g_x, size);
    hipMalloc((void **) &g_y, size);
    hipMalloc((void **) &g_z, size);

    hipMemcpy(g_x, &x, size, hipMemcpyHostToDevice);
    hipMemcpy(g_y, &y, size, hipMemcpyHostToDevice);


    mul<<<1,1>>>(g_x, g_y, g_z);
    hipMemcpy(&z, g_z, size, hipMemcpyDeviceToHost);
    printf ("%d * %d = %d\n", x, y, z);

    //Clean up
    hipFree(g_x);
    hipFree(g_y);
    hipFree(g_z);
    return 0;
}